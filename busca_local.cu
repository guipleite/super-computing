#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <iomanip>      
#include <vector>
#include <chrono> 
#include <sstream>
#include <string>
#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <thrust/random.h>
#include <thrust/random/uniform_real_distribution.h>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>

struct escolhe_alunos {
    int n_alunos,index,n_choices,seed,n_projetos;
    int *prefs,*aluno_projeto;

    escolhe_alunos (int *prefs,int *aluno_projeto,int n_projetos,int n_alunos,int n_choices,int seed): prefs(prefs),
                                                                               aluno_projeto(aluno_projeto),
                                                                               n_projetos(n_projetos),
                                                                               n_alunos(n_alunos),
                                                                               n_choices(n_choices),
                                                                               seed(seed)
                                                                               {};

    __device__ __host__
    int operator()(const int &i) {
      int choice_a1,choice_a2;

      int t=0;
      int melhor = 0;
      int satisfacao_atual= 0;
      bool flag =true;

      thrust::default_random_engine rng(i+26);
      thrust::uniform_int_distribution<int> distr(0,n_alunos-1);
      
      for(int p =n_alunos*i;p<(n_alunos*(i+1));p+=n_projetos){
        for (int j = 0; j < n_projetos; j++){
          aluno_projeto[p+j]= j;
        }
        t++;      
      }
      aluno_projeto[n_alunos*i]=0;

      for(int j=n_alunos*i;j<(n_alunos*(i+1));j++){
        index = distr(rng)+n_alunos*i;

        choice_a1 = aluno_projeto[j];
        choice_a2 = aluno_projeto[index];

        aluno_projeto[j] = choice_a2;
        aluno_projeto[index] = choice_a1;
      }

      for(int j=0;j<n_alunos;j++){
        satisfacao_atual+=prefs[(j*n_projetos)+aluno_projeto[j+n_alunos*i]];
      }

      melhor=satisfacao_atual;

      while (flag){
        flag = false;
      
        for(int j=0;j<n_alunos;j++){

          for(int k = 0; k<n_alunos;k++){
            satisfacao_atual = melhor;
            choice_a2 = aluno_projeto[k+n_alunos*i];
            choice_a1 = aluno_projeto[j+n_alunos*i];
            
            if(choice_a1!=choice_a2 and j!=k){
              satisfacao_atual-=prefs[(j*n_projetos)+choice_a1];
              satisfacao_atual-=prefs[(k*n_projetos)+choice_a2];

              satisfacao_atual+=prefs[(j*n_projetos)+choice_a2];
              satisfacao_atual+=prefs[(k*n_projetos)+choice_a1];

              if(satisfacao_atual>melhor){
                aluno_projeto[j+n_alunos*i] = choice_a2;
                aluno_projeto[k+n_alunos*i] = choice_a1;   
                melhor = satisfacao_atual;
                
                flag = true;
              }
            }
          }
        }
      }
      return  melhor;
    }
};

int main(){
    auto begin = std::chrono::high_resolution_clock::now();    

    int n,n_alunos,n_projetos,n_choices;
    thrust::device_vector<int> input, projs; //host?

    int iter = 0;
    int seed = 0;
    int c = 0;
    int melhor = 0;
    int in,ln;

    std::string p;

    if(const char* aaenv = std::getenv("ITER"))
      iter = atoi(aaenv);
    
    else 
      iter = 100000;

    if(const char* aenv = std::getenv("SEED"))
      seed = atoi(aenv);
    
    else 
      seed = 0;

    getline( std::cin, p);
    std::istringstream ss(p);

    while(ss >> n) {
      input.push_back(n);
    }
    
    n_alunos = input[0];
    n_projetos = input[1];
    n_choices = input[2];

    thrust::device_vector<int> prefs(n_alunos*n_projetos, -1);
    thrust::device_vector<int> aluno_projeto(n_alunos*iter, -1); // não escolheu projeto ainda
    thrust::device_vector<int> sat(iter, -1);
    int head = 0;

    for (int i=0; i< n_alunos; i++){        
        getline(std::cin, p);
        std::istringstream ss(p);
        projs.clear();

        while(ss >> n) {
            projs.push_back(n);
        }

        for(int j = 0; j < n_choices; j++){           
            prefs[(i*n_projetos)+projs[j]] = pow(n_choices - j, 2);
            head++;
        }
    }

    thrust::counting_iterator<int> i(0);
    escolhe_alunos refs(thrust::raw_pointer_cast(prefs.data()),
                        thrust::raw_pointer_cast(aluno_projeto.data()),
                        n_projetos,n_alunos,n_choices,seed);
                        
    thrust::transform(i, i+sat.size(), sat.begin(),refs);

    // Para roddar os testes do Jupyter, comente as linhas abaixo
    for (int i=0; i<iter; i++){
      if(sat[i]>melhor){
        melhor = sat[i];
        in =i;
      }
    } 
  //Para rodar o Jupyter comentar a parte a seguir:
    std::cout  <<melhor<< " 1\n";

    for (auto i=aluno_projeto.begin(); i!= aluno_projeto.end(); i++){
      if(c%n_alunos==0 and c!=0){
        ln++;
      }

      if(ln==in){
        std::cout << *i <<" " ;
        
      }
      c++;
    }
    std::cerr << "\n";

    auto end = std::chrono::high_resolution_clock::now();    
    auto dur = end - begin;
    auto ms = std::chrono::duration_cast<std::chrono::microseconds>(dur).count();
    //Para rodar o Jupyter descomentar a parte a seguir:
    // std::cout << "\n" << ms;}

